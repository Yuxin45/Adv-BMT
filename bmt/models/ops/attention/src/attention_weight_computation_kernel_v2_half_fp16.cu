#include "hip/hip_runtime.h"
/*
Transformer function helper function.
Written by tomztyang,
2021/08/23
*/

#include <math.h>
#include <stdio.h>
#include <torch/extension.h>
#include <hip/hip_fp16.h>
#include <ATen/native/cuda/KernelUtils.cuh>

#define THREADS_PER_BLOCK 256
#define DIVUP(m,n) ((m) / (n) + ((m) % (n) > 0))
// #define DEBUG


template <unsigned int d>
__global__ void attention_weight_computation_forward_v2_fp16(
    int b, int total_query_num, int local_size,
    int total_key_num, int nhead, int hdim,
    const int *query_batch_cnt, const int *key_batch_cnt, const int* index_pair_batch,
    const int *index_pair,
    const at::Half *query_features, const at::Half* key_features,
    at::Half *output) {
    // dim3 blocks(total_query_num, nhead); dim3 threads(local_size);
    // params query_batch_cnt: [b]
    // params key_batch_cnt: [b]
    // params index_pair_batch: [total_query_num]
    // params index_pair: [total_query_num, local_size]
    // params query_features: [total_query_num, nhead, hdim]
    // params key_features: [total_key_num, nhead, hdim]
    // params output: [total_query_num, local_size, nhead]

    int query_idx = blockIdx.x;
    int head_idx = blockIdx.y;
    int local_key_idx = threadIdx.x;

    int index = query_idx * local_size + local_key_idx;
    if (query_idx >= total_query_num ||
        head_idx >= nhead ||
        local_key_idx >= local_size) return;

    // build shared query features.
    __shared__ at::Half shared_query_features[d];
    for (int i = local_key_idx; i < hdim; i += blockDim.x){
        shared_query_features[i] = query_features[
            query_idx * nhead * hdim + head_idx * hdim + i];
    }
    __syncthreads();

    if (index_pair[index] == -1){
        // Ignore index.
        return;
    }

    // get real key_idx.
    int batch_idx = index_pair_batch[query_idx];
    int key_start_idx = 0;
    for (int i = 0; i < batch_idx; i++){
        key_start_idx += key_batch_cnt[i];
    }
    key_start_idx += index_pair[index];

    // get key features.
    key_features += key_start_idx * nhead * hdim + head_idx * hdim;
    output += index * nhead + head_idx;

    at::Half attn_weight = __int2half_rn(0);
    for (int i = 0; i < hdim; i++){
        attn_weight = __hadd(attn_weight, __hmul(key_features[i], shared_query_features[i]));
    }
    output[0] = attn_weight;
}


void attention_weight_computation_launcher_v2_fp16(
    int b, int total_query_num, int local_size,
    int total_key_num, int nhead, int hdim,
    const int *query_batch_cnt, const int *key_batch_cnt, const int* index_pair_batch,
    const int *index_pair,
    const at::Half *query_features, const at::Half* key_features,
    at::Half *output){
    // params query_batch_cnt: [b]
    // params key_batch_cnt: [b]
    // params index_pair_batch: [total_query_num]
    // params index_pair: [total_query_num, local_size]
    // params query_features: [total_query_num, nhead, hdim]
    // params key_features: [total_key_num, nhead, hdim]
    // params output: [total_query_num, local_size, nhead]
    if (hdim > 150){
        throw "hdim should be <= 150.";
    }

    dim3 blocks(total_query_num, nhead);
    dim3 threads(local_size);
    switch(hdim){  // switch hdim for utilizing different shared vectors.
        case 16:
            attention_weight_computation_forward_v2_fp16<16><<<blocks, threads>>>(
                b, total_query_num, local_size, total_key_num, nhead, hdim,
                query_batch_cnt, key_batch_cnt, index_pair_batch,
                index_pair, query_features, key_features,
                output);
            break;
        case 24:
            attention_weight_computation_forward_v2_fp16<24><<<blocks, threads>>>(
                b, total_query_num, local_size, total_key_num, nhead, hdim,
                query_batch_cnt, key_batch_cnt, index_pair_batch,
                index_pair, query_features, key_features,
                output);
            break;
        case 32:
            attention_weight_computation_forward_v2_fp16<32><<<blocks, threads>>>(
                b, total_query_num, local_size, total_key_num, nhead, hdim,
                query_batch_cnt, key_batch_cnt, index_pair_batch,
                index_pair, query_features, key_features,
                output);
            break;
        case 48:
            attention_weight_computation_forward_v2_fp16<48><<<blocks, threads>>>(
                b, total_query_num, local_size, total_key_num, nhead, hdim,
                query_batch_cnt, key_batch_cnt, index_pair_batch,
                index_pair, query_features, key_features,
                output);
            break;
        case 64:
            attention_weight_computation_forward_v2_fp16<64><<<blocks, threads>>>(
                b, total_query_num, local_size, total_key_num, nhead, hdim,
                query_batch_cnt, key_batch_cnt, index_pair_batch,
                index_pair, query_features, key_features,
                output);
            break;
        case 128:
            attention_weight_computation_forward_v2_fp16<128><<<blocks, threads>>>(
                b, total_query_num, local_size, total_key_num, nhead, hdim,
                query_batch_cnt, key_batch_cnt, index_pair_batch,
                index_pair, query_features, key_features,
                output);
            break;
        default:
            attention_weight_computation_forward_v2_fp16<150><<<blocks, threads>>>(
                b, total_query_num, local_size, total_key_num, nhead, hdim,
                query_batch_cnt, key_batch_cnt, index_pair_batch,
                index_pair, query_features, key_features,
                output);
            break;
    }
}


template <unsigned int d>
__global__ void attention_weight_computation_backward_v2_fp16(
    int b, int total_query_num, int local_size,
    int total_key_num, int nhead, int hdim,
    const int *query_batch_cnt, const int *key_batch_cnt, const int* index_pair_batch,
    const int *index_pair,
    const at::Half *query_features, const at::Half* key_features,
    at::Half *grad_out, at::Half * grad_query_features, at::Half * grad_key_features) {
    // dim3 blocks(total_query_num, nhead); dim3 threads(local_size);
    // params query_batch_cnt: [b]
    // params key_batch_cnt: [b]
    // params index_pair_batch: [total_query_num]
    // params index_pair: [total_query_num, local_size]
    // params query_features: [total_query_num, nhead, hdim]
    // params key_features: [total_key_num, nhead, hdim]
    // params grad_out: [total_query_num, local_size, nhead]
    // params grad_query_features: [total_query_num, nhead, hdim]
    // params grad_key_features: [total_key_num, nhead, hdim]

    int query_idx = blockIdx.x;
    int head_idx = blockIdx.y;
    int local_key_idx = threadIdx.x;
    int index = query_idx * local_size + local_key_idx;

    if (query_idx >= total_query_num ||
        head_idx >= nhead ||
        local_key_idx >= local_size) return;

    // build shared query features.
    __shared__ at::Half shared_query_features[d];
    __shared__ at::Half shared_grad_query_features[d];
    for (int i = local_key_idx; i < hdim; i += blockDim.x){
        shared_query_features[i] = query_features[
            query_idx * nhead * hdim + head_idx * hdim + i];
        shared_grad_query_features[i] = __int2half_rn(0);
    }
    __syncthreads();

    if (index_pair[index] != -1){
        int batch_idx = index_pair_batch[query_idx];
        int key_start_idx = 0;
        for (int i = 0; i < batch_idx; i++){
            key_start_idx += key_batch_cnt[i];
        }
        key_start_idx += index_pair[index];

        key_features += key_start_idx * nhead * hdim + head_idx * hdim;
        grad_key_features += key_start_idx * nhead * hdim + head_idx * hdim;

        at::Half gradient = grad_out[index * nhead + head_idx];
        for (int i = 0; i < hdim; i++){
//            atomicAdd(
//                shared_grad_query_features + i,
//                gradient * key_features[i]);
//            atomicAdd(
//                grad_key_features + i,
//                gradient * shared_query_features[i]);
            at::native::fastAtomicAdd(
                shared_grad_query_features + i,
                0, 0,
                gradient * key_features[i], true);
            at::native::fastAtomicAdd(
                grad_key_features + i, 0, 0,
                gradient * shared_query_features[i], true);
        }
    }
    __syncthreads();

    grad_query_features += query_idx * nhead * hdim + head_idx * hdim;
    for (int i = local_key_idx; i < hdim; i += blockDim.x){
        grad_query_features[i] = shared_grad_query_features[i];
    }
}


void attention_weight_computation_grad_launcher_v2_fp16(
    int b, int total_query_num, int local_size,
    int total_key_num, int nhead, int hdim,
    const int *query_batch_cnt, const int *key_batch_cnt, const int* index_pair_batch,
    const int *index_pair,
    const at::Half *query_features, const at::Half* key_features,
    at::Half *grad_out, at::Half* grad_query_features, at::Half* grad_key_features){
    // params query_batch_cnt: [b]
    // params key_batch_cnt: [b]
    // params index_pair_batch: [total_query_num]
    // params index_pair: [total_query_num, local_size]
    // params query_features: [total_query_num, nhead, hdim]
    // params key_features: [total_key_num, nhead, hdim]
    // params grad_out: [total_query_num, local_size, nhead]
    // params grad_query_features: [total_query_num, nhead, hdim]
    // params grad_key_features: [total_key_num, nhead, hdim]
    if (hdim > 150){
        throw "hdim should be <= 150.";
    }

    dim3 blocks(total_query_num, nhead);
    dim3 threads(local_size);

    switch(hdim){
        case 16:
            attention_weight_computation_backward_v2_fp16<16><<<blocks, threads>>>(
                b, total_query_num, local_size, total_key_num, nhead, hdim,
                query_batch_cnt, key_batch_cnt, index_pair_batch,
                index_pair, query_features, key_features,
                grad_out, grad_query_features, grad_key_features);
            break;
        case 24:
            attention_weight_computation_backward_v2_fp16<24><<<blocks, threads>>>(
                b, total_query_num, local_size, total_key_num, nhead, hdim,
                query_batch_cnt, key_batch_cnt, index_pair_batch,
                index_pair, query_features, key_features,
                grad_out, grad_query_features, grad_key_features);
            break;
        case 32:
            attention_weight_computation_backward_v2_fp16<32><<<blocks, threads>>>(
                b, total_query_num, local_size, total_key_num, nhead, hdim,
                query_batch_cnt, key_batch_cnt, index_pair_batch,
                index_pair, query_features, key_features,
                grad_out, grad_query_features, grad_key_features);
            break;
        case 48:
            attention_weight_computation_backward_v2_fp16<48><<<blocks, threads>>>(
                b, total_query_num, local_size, total_key_num, nhead, hdim,
                query_batch_cnt, key_batch_cnt, index_pair_batch,
                index_pair, query_features, key_features,
                grad_out, grad_query_features, grad_key_features);
            break;
        case 64:
            attention_weight_computation_backward_v2_fp16<64><<<blocks, threads>>>(
                b, total_query_num, local_size, total_key_num, nhead, hdim,
                query_batch_cnt, key_batch_cnt, index_pair_batch,
                index_pair, query_features, key_features,
                grad_out, grad_query_features, grad_key_features);
            break;
        case 128:
            attention_weight_computation_backward_v2_fp16<128><<<blocks, threads>>>(
                b, total_query_num, local_size, total_key_num, nhead, hdim,
                query_batch_cnt, key_batch_cnt, index_pair_batch,
                index_pair, query_features, key_features,
                grad_out, grad_query_features, grad_key_features);
            break;
        default:
            attention_weight_computation_backward_v2_fp16<150><<<blocks, threads>>>(
                b, total_query_num, local_size, total_key_num, nhead, hdim,
                query_batch_cnt, key_batch_cnt, index_pair_batch,
                index_pair, query_features, key_features,
                grad_out, grad_query_features, grad_key_features);
            break;
    }
}