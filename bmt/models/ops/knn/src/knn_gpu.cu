#include "hip/hip_runtime.h"
// Motion Transformer (MTR): https://arxiv.org/abs/2209.13508
// Published at NeurIPS 2022
// Written by Li Jiang, Shaoshuai Shi 
// All Rights Reserved


#include "knn_gpu.h"

#include <stdio.h>
#include <stdlib.h>
#include <torch/extension.h>
#include <hip/hip_bf16.h>
#include <hip/hip_fp16.h>

#define THREADS_PER_BLOCK 256
#define DIVUP(m,n) ((m) / (n) + ((m) % (n) > 0))

__global__ void knn_batch_cuda_(int n, int m, int k, const float *__restrict__ xyz, const float *__restrict__ query_xyz, const int *__restrict__ batch_idxs, const int *__restrict__ query_batch_offsets, int *__restrict__ idx) {
    int pt_idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (pt_idx >= n) return;

    xyz += pt_idx * 3;
    idx += pt_idx * k;

    float ox = xyz[0];
    float oy = xyz[1];
    float oz = xyz[2];

    float best[100];
    int besti[100];
    for(int i = 0; i < k; i++){
        best[i] = 1e20;
        besti[i] = -1;
    }

    int batch_idx = batch_idxs[pt_idx];
    int start = query_batch_offsets[batch_idx];
    int end = query_batch_offsets[batch_idx + 1];

    for (int i = start; i < end; ++i) {
        float x = query_xyz[i * 3 + 0];
        float y = query_xyz[i * 3 + 1];
        float z = query_xyz[i * 3 + 2];
        float d2 = (ox - x) * (ox - x) + (oy - y) * (oy - y) + (oz - z) * (oz - z);
        for(int p = 0; p < k; p++){
            if(d2 < best[p]){
                for(int q = k - 1; q > p; q--){
                    best[q] = best[q - 1];
                    besti[q] = besti[q - 1];
                }
                best[p] = d2;
                besti[p] = i - start;
                break;
            }
        }
    }

    for(int i = 0; i < k; i++){
        idx[i] = besti[i];
    }
}


__global__ void knn_batch_mlogk_cuda_(int n, int m, int k, const float *__restrict__ xyz, const float *__restrict__ query_xyz, const int *__restrict__ batch_idxs, const int *__restrict__ query_batch_offsets, int *__restrict__ idx) {
    int pt_idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (pt_idx >= n) return;

    xyz += pt_idx * 3;
    idx += pt_idx * k;

    float ox = xyz[0];
    float oy = xyz[1];
    float oz = xyz[2];

    float best[150];
    int besti[150];

    int heap_len = 0;

    for(int i = 0; i <= k; i++){
        best[i] = std::numeric_limits<float>::infinity();
        besti[i] = -1;
    }

    int batch_idx = batch_idxs[pt_idx];
    int start = query_batch_offsets[batch_idx];
    int end = query_batch_offsets[batch_idx + 1];
    int temp_i;
    float temp_f;

    for (int i = start; i < end; ++i) {
        float x = query_xyz[i * 3 + 0];
        float y = query_xyz[i * 3 + 1];
        float z = query_xyz[i * 3 + 2];
        float d2 = (ox - x) * (ox - x) + (oy - y) * (oy - y) + (oz - z) * (oz - z);

        if (heap_len < k){
            heap_len++;
            best[heap_len] = d2;
            besti[heap_len] = i - start;
            int cur_idx = heap_len, fa_idx = cur_idx >> 1;

            while (fa_idx > 0){
                if (best[cur_idx] < best[fa_idx]) break;

                temp_i = besti[cur_idx]; besti[cur_idx] = besti[fa_idx]; besti[fa_idx] = temp_i; 
                temp_f = best[cur_idx]; best[cur_idx] = best[fa_idx]; best[fa_idx] = temp_f;
                cur_idx = fa_idx;
                fa_idx = cur_idx >> 1;
            }
        }
        else{
            if (d2 > best[1]) continue;
            best[1] = d2; besti[1] = i - start;

            int cur_idx = 1, son_idx;
            while (cur_idx <= k){
                son_idx = cur_idx << 1;
                if (son_idx > k) break;
                if (son_idx + 1 <= k && best[son_idx] < best[son_idx + 1]){
                    son_idx++;
                }

                if (son_idx <= k && best[cur_idx] < best[son_idx]){
                    temp_i = besti[cur_idx]; besti[cur_idx] = besti[son_idx]; besti[son_idx] = temp_i; 
                    temp_f = best[cur_idx]; best[cur_idx] = best[son_idx]; best[son_idx] = temp_f;
                }
                else break;
                cur_idx = son_idx;
            }
        }
    }
    
    for(int i = 1; i <= k; i++){
        idx[i - 1] = besti[i];
    }
    // delete [] best;
    // delete [] besti;
}



__global__ void knn_batch_mlogk_cuda_half_(int n, int m, int k, const at::BFloat16 *__restrict__ xyz, const at::BFloat16 *__restrict__ query_xyz, const int *__restrict__ batch_idxs, const int *__restrict__ query_batch_offsets, int *__restrict__ idx) {
    int pt_idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (pt_idx >= n) return;

    xyz += pt_idx * 3;
    idx += pt_idx * k;

    at::BFloat16 ox = xyz[0];
    at::BFloat16 oy = xyz[1];
    at::BFloat16 oz = xyz[2];

    at::BFloat16 best[150];
    int besti[150];

    int heap_len = 0;

    for(int i = 0; i <= k; i++){
        best[i] = __float2bfloat16(std::numeric_limits<float>::infinity());
        besti[i] = -1;
    }

    int batch_idx = batch_idxs[pt_idx];
    int start = query_batch_offsets[batch_idx];
    int end = query_batch_offsets[batch_idx + 1];
    int temp_i;
    at::BFloat16 temp_f;

    for (int i = start; i < end; ++i) {
        at::BFloat16 x = query_xyz[i * 3 + 0];
        at::BFloat16 y = query_xyz[i * 3 + 1];
        at::BFloat16 z = query_xyz[i * 3 + 2];
        at::BFloat16 d2 = (ox - x) * (ox - x) + (oy - y) * (oy - y) + (oz - z) * (oz - z);

        if (heap_len < k){
            heap_len++;
            best[heap_len] = d2;
            besti[heap_len] = i - start;
            int cur_idx = heap_len, fa_idx = cur_idx >> 1;

            while (fa_idx > 0){
                if (best[cur_idx] < best[fa_idx]) break;

                temp_i = besti[cur_idx]; besti[cur_idx] = besti[fa_idx]; besti[fa_idx] = temp_i;
                temp_f = best[cur_idx]; best[cur_idx] = best[fa_idx]; best[fa_idx] = temp_f;
                cur_idx = fa_idx;
                fa_idx = cur_idx >> 1;
            }
        }
        else{
            if (d2 > best[1]) continue;
            best[1] = d2; besti[1] = i - start;

            int cur_idx = 1, son_idx;
            while (cur_idx <= k){
                son_idx = cur_idx << 1;
                if (son_idx > k) break;
                if (son_idx + 1 <= k && best[son_idx] < best[son_idx + 1]){
                    son_idx++;
                }

                if (son_idx <= k && best[cur_idx] < best[son_idx]){
                    temp_i = besti[cur_idx]; besti[cur_idx] = besti[son_idx]; besti[son_idx] = temp_i; 
                    temp_f = best[cur_idx]; best[cur_idx] = best[son_idx]; best[son_idx] = temp_f;
                }
                else break;
                cur_idx = son_idx;
            }
        }
    }
    
    for(int i = 1; i <= k; i++){
        idx[i - 1] = besti[i];
    }
    // delete [] best;
    // delete [] besti;
}




__global__ void knn_batch_mlogk_cuda_half_fp16_(int n, int m, int k, const at::Half *__restrict__ xyz, const at::Half *__restrict__ query_xyz, const int *__restrict__ batch_idxs, const int *__restrict__ query_batch_offsets, int *__restrict__ idx) {
    int pt_idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (pt_idx >= n) return;

    xyz += pt_idx * 3;
    idx += pt_idx * k;

    at::Half ox = xyz[0];
    at::Half oy = xyz[1];
    at::Half oz = xyz[2];

    at::Half best[150];
    int besti[150];

    int heap_len = 0;

    for(int i = 0; i <= k; i++){
        best[i] = __float2half(std::numeric_limits<float>::infinity());
        besti[i] = -1;
    }

    int batch_idx = batch_idxs[pt_idx];
    int start = query_batch_offsets[batch_idx];
    int end = query_batch_offsets[batch_idx + 1];
    int temp_i;
    at::Half temp_f;

    for (int i = start; i < end; ++i) {
        at::Half x = query_xyz[i * 3 + 0];
        at::Half y = query_xyz[i * 3 + 1];
        at::Half z = query_xyz[i * 3 + 2];
        at::Half d2 = (ox - x) * (ox - x) + (oy - y) * (oy - y) + (oz - z) * (oz - z);

        if (heap_len < k){
            heap_len++;
            best[heap_len] = d2;
            besti[heap_len] = i - start;
            int cur_idx = heap_len, fa_idx = cur_idx >> 1;

            while (fa_idx > 0){
                if (best[cur_idx] < best[fa_idx]) break;

                temp_i = besti[cur_idx]; besti[cur_idx] = besti[fa_idx]; besti[fa_idx] = temp_i;
                temp_f = best[cur_idx]; best[cur_idx] = best[fa_idx]; best[fa_idx] = temp_f;
                cur_idx = fa_idx;
                fa_idx = cur_idx >> 1;
            }
        }
        else{
            if (d2 > best[1]) continue;
            best[1] = d2; besti[1] = i - start;

            int cur_idx = 1, son_idx;
            while (cur_idx <= k){
                son_idx = cur_idx << 1;
                if (son_idx > k) break;
                if (son_idx + 1 <= k && best[son_idx] < best[son_idx + 1]){
                    son_idx++;
                }

                if (son_idx <= k && best[cur_idx] < best[son_idx]){
                    temp_i = besti[cur_idx]; besti[cur_idx] = besti[son_idx]; besti[son_idx] = temp_i; 
                    temp_f = best[cur_idx]; best[cur_idx] = best[son_idx]; best[son_idx] = temp_f;
                }
                else break;
                cur_idx = son_idx;
            }
        }
    }
    
    for(int i = 1; i <= k; i++){
        idx[i - 1] = besti[i];
    }
    // delete [] best;
    // delete [] besti;
}




void knn_batch_cuda(int n, int m, int k, const float *xyz, const float *query_xyz, const int *batch_idxs, const int *query_batch_offsets, int *idx, hipStream_t stream) {
    // param xyz: (n, 3), float
    // param query_xyz: (m, 3), float
    // param batch_idxs: (n), int
    // param query_batch_offsets: (B + 1), int, offsets[-1] = m
    // param idx: (n, k), int

    hipError_t err;

    dim3 blocks(DIVUP(n, THREADS_PER_BLOCK));  // blockIdx.x(col), blockIdx.y(row)
    dim3 threads(THREADS_PER_BLOCK);

    knn_batch_cuda_<<<blocks, threads, 0, stream>>>(n, m, k, xyz, query_xyz, batch_idxs, query_batch_offsets, idx);
    // hipDeviceSynchronize();  // for using printf in kernel function

    err = hipGetLastError();
    if (hipSuccess != err) {
        fprintf(stderr, "CUDA kernel failed : %s\n", hipGetErrorString(err));
        exit(-1);
    }
}


void knn_batch_mlogk_cuda(int n, int m, int k, const float *xyz, const float *query_xyz, const int *batch_idxs, const int *query_batch_offsets, int *idx, hipStream_t stream) {
    // param xyz: (n, 3), float
    // param query_xyz: (m, 3), float
    // param batch_idxs: (n), int
    // param query_batch_offsets: (B + 1), int, offsets[-1] = m
    // param idx: (n, k), int

    hipError_t err;

    dim3 blocks(DIVUP(n, THREADS_PER_BLOCK));  // blockIdx.x(col), blockIdx.y(row)
    dim3 threads(THREADS_PER_BLOCK);

    knn_batch_mlogk_cuda_<<<blocks, threads, 0, stream>>>(n, m, k, xyz, query_xyz, batch_idxs, query_batch_offsets, idx);
    // hipDeviceSynchronize();  // for using printf in kernel function

    err = hipGetLastError();
    if (hipSuccess != err) {
        fprintf(stderr, "CUDA kernel failed : %s\n", hipGetErrorString(err));
        exit(-1);
    }
}


void knn_batch_mlogk_cuda_half(int n, int m, int k, const at::BFloat16 *xyz, const at::BFloat16 *query_xyz, const int *batch_idxs, const int *query_batch_offsets, int *idx, hipStream_t stream) {
    // param xyz: (n, 3), at::BFloat16
    // param query_xyz: (m, 3), at::BFloat16
    // param batch_idxs: (n), int
    // param query_batch_offsets: (B + 1), int, offsets[-1] = m
    // param idx: (n, k), int

    hipError_t err;

    dim3 blocks(DIVUP(n, THREADS_PER_BLOCK));  // blockIdx.x(col), blockIdx.y(row)
    dim3 threads(THREADS_PER_BLOCK);

    knn_batch_mlogk_cuda_half_<<<blocks, threads, 0, stream>>>(n, m, k, xyz, query_xyz, batch_idxs, query_batch_offsets, idx);
    // hipDeviceSynchronize();  // for using printf in kernel function

    err = hipGetLastError();
    if (hipSuccess != err) {
        fprintf(stderr, "CUDA kernel failed : %s\n", hipGetErrorString(err));
        exit(-1);
    }
}


void knn_batch_mlogk_cuda_half_fp16(int n, int m, int k, const at::Half *xyz, const at::Half *query_xyz, const int *batch_idxs, const int *query_batch_offsets, int *idx, hipStream_t stream) {
    // param xyz: (n, 3), at::Half
    // param query_xyz: (m, 3), at::Half
    // param batch_idxs: (n), int
    // param query_batch_offsets: (B + 1), int, offsets[-1] = m
    // param idx: (n, k), int

    hipError_t err;

    dim3 blocks(DIVUP(n, THREADS_PER_BLOCK));  // blockIdx.x(col), blockIdx.y(row)
    dim3 threads(THREADS_PER_BLOCK);

    knn_batch_mlogk_cuda_half_fp16_<<<blocks, threads, 0, stream>>>(n, m, k, xyz, query_xyz, batch_idxs, query_batch_offsets, idx);
    // hipDeviceSynchronize();  // for using printf in kernel function

    err = hipGetLastError();
    if (hipSuccess != err) {
        fprintf(stderr, "CUDA kernel failed : %s\n", hipGetErrorString(err));
        exit(-1);
    }
}
